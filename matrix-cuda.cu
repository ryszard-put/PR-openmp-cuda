
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void gpu_matrixMult(int* A, int* B, int* C, int N)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int sum = 0;
  int row_n = row * N;

  if(row < N && col < N){
    for(int i = 0; i < N; i++){
      sum += A[row_n + i] * B[i * N + col];
    }
  }
  C[row_n + col] = sum;
}

void matrixMultiplication(int *A, int *B, int *C, int N){

    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
        if (N*N > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
        }

    gpu_matrixMult<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}

__host__ void cpu_matrixMul(int* A, int* B, int* result, int N)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      int sum = 0;
      for(int k = 0; k < N; k++) {
        sum += A[N * i + k] + B[N * k + j];
      }
      result[N * i + j] = sum;
    }
  }
}

int main(int argc, char** argv)
{
  srand(time(NULL));
  if (argc < 2) {
    perror("Podaj wymiar macierzy");
    exit(-1);
  }

  int size = atoi(argv[1]);
  int* h_matrix = (int*)malloc(sizeof(int) * size * size);
  int* h_result = (int*)malloc(sizeof(int) * size * size);

  for(int i = 0; i < size * size; i++) {
    h_matrix[i] = rand() % 5 + 1;
  }

  int *d_matrix, *d_result;
  hipMalloc((void**)&d_matrix, sizeof(int) * size * size);
  hipMalloc((void**)&d_result, sizeof(int) * size * size);
  hipMemcpy(d_matrix, h_matrix, sizeof(int) * size * size, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  matrixMultiplication(d_matrix, d_matrix, d_result, size);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  hipMemcpy(h_result, d_result, sizeof(int) * size * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  printf("Rozmiar macierzy: %dx%d, czas CUDA: %f\n", size, size, time);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // testowanie na cpu
  int* result_cpu = (int*)malloc(sizeof(int) * size * size);

  clock_t start_cpu = clock();
  cpu_matrixMul(h_matrix, h_matrix, result_cpu, size);
  clock_t end_cpu = clock();

  double time_cpu = (double)(end_cpu - start_cpu) / (double)CLOCKS_PER_SEC;
  printf("Rozmiar macierzy: %dx%d, czas CPU: %f\n", size, size, time_cpu);
  hipFree(d_matrix);
  hipFree(d_result);
}